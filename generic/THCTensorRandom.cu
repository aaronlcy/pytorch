
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorRandom.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)
THC_API void THCTensor_(uniform)(THCState* state, THCTensor *self_, double a, double b)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_uniform<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, a, b);

  THCTensor_(freeCopyTo)(state, self, self_);
};
#undef NUM_BLOCKS

THC_API void THCTensor_(rand)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(uniform)(state, r_, 0, 1);
}

#endif

#endif
